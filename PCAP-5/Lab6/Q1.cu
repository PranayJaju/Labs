#include<stdio.h>
#include<stdlib.h>
#include <string.h>
#include<hip/hip_runtime.h>

#define MAX_SEN_LEN 1024
#define MAX_WORD_LEN 64
#define MAX_BLOCKS 1024
#define THREADS_PER_BLOCK 1024

__global__ void countWordOccurrences(char *sen, char *word, int *count, int senLen, int wordLen)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while(idx < senLen)
    {
        int match = 1;
        for (int i = 0; i < wordLen; i++)
            if (sen[idx + i] != word[i])
            {
                match = 0;
                break;
            }

        if (match == 1)
        {
            if((sen[idx+wordLen] == ' ' || idx == senLen-wordLen) && sen[idx-1] == ' ')
            	atomicAdd(count, 1);
            idx += wordLen;
        } 
        else
            idx += blockDim.x;
    }
}

int main() {
    char sen[MAX_SEN_LEN];
    char word[MAX_WORD_LEN];
    int count = 0;
    
    printf("Enter a sen: ");
    fgets(sen, sizeof(sen), stdin);
    sen[strlen(sen)-1] = '\0';
    
    printf("Enter the target word to count: ");
    scanf("%s", word);
    
    int senLen = strlen(sen);
    int wordLen = strlen(word);
    
    char *d_sen, *d_word;
    int *d_count;
    
    hipMalloc((void**)&d_sen, senLen);
    hipMalloc((void**)&d_word, wordLen);
    hipMalloc((void**)&d_count, sizeof(int));
    
    hipMemcpy(d_sen, sen, senLen, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLen, hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    
    int numBlocks = (senLen + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    countWordOccurrences<<<numBlocks, THREADS_PER_BLOCK>>>(d_sen, d_word, d_count, senLen, wordLen);
    
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("The word '%s' appeared %d times in the sen.\n", word, count);
    
    hipFree(d_sen);
    hipFree(d_word);
    hipFree(d_count);
}