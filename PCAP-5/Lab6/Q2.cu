#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_STR_LEN 1024

__global__ void SRS(char *S, char *RS, int strLen)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= strLen) return;

    int startpos = 0;

    for(int i = 0; i < tid; i++) startpos += strLen - i;

    for(int i = startpos, j=0; j < strLen - tid; i++, j++)
        RS[i] = S[j];
}

int main() {
    char S[MAX_STR_LEN];
    char RS[MAX_STR_LEN];
    
    printf("Enter a string: ");
    fgets(S, MAX_STR_LEN, stdin);
    S[strlen(S)-1] = '\0';

    int strLen = strlen(S);
    int size = MAX_STR_LEN * sizeof(char);
    
    char *d_S, *d_RS;
    
    hipMalloc((void**)&d_S, size);
    hipMalloc((void**)&d_RS, size);
    
    hipMemcpy(d_S, S, size, hipMemcpyHostToDevice);
    
    int numBlocks = (strLen + 255) / 256; // Adjust block size according to your needs
    SRS<<<numBlocks, 256>>>(d_S, d_RS, strLen);
    
    hipMemcpy(RS, d_RS, size, hipMemcpyDeviceToHost);
    
    //RS[strLen*(strLen-1)] = '\0'; // Null-terminate RS
    
    printf("S: %s\n", S);
    printf("RS: %s\n", RS);
    
    hipFree(d_S);
    hipFree(d_RS);
    
    return 0;
}