
#include <hip/hip_runtime.h>
#include<stdio.h>

enum method_type{ROW, COL, ELE};
enum operation{ADD, MULTIPLY};

__device__ void Add(int *a, int *b, int *c, int idx)
{
	c[idx] = a[idx] + b[idx];
}

__device__ void Multiply(int *a, int *b, int *c, int row, int col, int nc)
{
	int idx = row*nc + col;
	c[idx] = 0;
	for(int k = 0; k < nc; k++)
		c[idx] += a[row*nc + k] * b[k*nc + col];
}

__global__ void MatrixOp(int *a, int *b, int*c, int row, int col, operation op, method_type method)
{
	int R = blockDim.y*blockIdx.y + threadIdx.y;
	int C = blockDim.x*blockIdx.x + threadIdx.x;

	switch(method)
	{
		case ROW:
			if(R >= row) return;
			for(int i = 0; i < col; i++)
			{
				if(op == ADD)
					Add(a, b, c, R*col + i);
				else if(op == MULTIPLY)
					Multiply(a, b, c, R, i, col);
			}
			break;

		case COL:
			if(C >= col) return;
			for(int i = 0; i < row; i++)
			{
				if(op == ADD)
					Add(a, b, c, i*col + C);
				else if(op == MULTIPLY)
					Multiply(a, b, c, i, C, col);
			}
			break;

		case ELE:
			if(R >= row || C >= col) return;
			if(op == ADD)
				Add(a, b, c, R*col + C);
			else if(op == MULTIPLY)
				Multiply(a, b, c, R, C, col);
			break;

		default:
			break;
	}
}

int main()
{
	int a[3][3] = {{1,2,3},{4,5,6},{7,8,9}};
	int b[3][3] = {{1,0,0},{0,1,0},{0,0,1}};
	int c[3][3];

	int *da, *db, *dc;
	int size = 3*3*sizeof(int);

	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size);

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, size, hipMemcpyHostToDevice);

	dim3 grid(1,1,1);
	dim3 block(5,5,1);

	MatrixOp<<<grid, block>>>(da, db, dc, 3, 3, ADD, ELE);

	hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < 3; i++)
	{
		for(int j = 0; j < 3; j++)
			printf("%d  ", c[i][j]);
		printf("\n");
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}