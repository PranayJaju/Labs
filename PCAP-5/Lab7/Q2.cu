
#include <hip/hip_runtime.h>
#include<stdio.h>

enum method_type{ROW, COL, ELE};

__global__ void MatrixMul(int *a, int *b, int *c, int row, int col, method_type method)
{
	int R = blockDim.y*blockIdx.y + threadIdx.y;
	int C = blockDim.x*blockIdx.x + threadIdx.x;

	switch(method)
	{
		case ROW:
			if(R >= row) return;
			for(int i = 0; i < col; i++)
			{
				int idx = R*col + i;
				c[idx] = 0;
				for(int k = 0; k < col; k++)
					c[idx] += a[R*col + k] * b[k*col + i];
			}
			break;

		case COL:
			if(C >= col) return;
			for(int i = 0; i < row; i++)
			{
				int idx = i*col + C;
				c[idx] = 0;
				for(int k = 0; k < row; k++)
					c[idx] += a[i*col + k] * b[k*col + C];
			}
			break;

		case ELE:
			if(R >= row || C >= col) return;
			c[R*col + C] = 0;
			for(int k = 0; k < col; k++)
				c[R*col + C] += a[R*col + k] * b[k*col + C];
			break;

		default:
			break;
	}
}

int main()
{
	int a[3][3] = {{1,2,3},{4,5,6},{7,8,9}};
	int b[3][3] = {{1,0,0},{0,1,0},{0,0,1}};
	int c[3][3];

	int *da, *db, *dc;
	int size = 3*3*sizeof(int);

	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size);

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, size, hipMemcpyHostToDevice);

	dim3 grid(1,1,1);
	dim3 block(5,5,1);

	MatrixMul<<<grid, block>>>(da, db, dc, 3, 3, ELE);

	hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < 3; i++)
	{
		for(int j = 0; j < 3; j++)
			printf("%d  ", c[i][j]);
		printf("\n");
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}