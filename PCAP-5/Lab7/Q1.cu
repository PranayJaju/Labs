
#include <hip/hip_runtime.h>
#include<stdio.h>

enum method_type{ROW, COL, ELE};

__global__ void MatrixAdd(int *a, int *b, int *c, int row, int col, method_type method)
{
	int R = blockDim.y*blockIdx.y + threadIdx.y;
	int C = blockDim.x*blockIdx.x + threadIdx.x;

	switch(method)
	{
		case ROW:
			if(R >= row) return;
			for(int i = 0; i < col; i++)
				c[R*col + i] = a[R*row + i] + b[R*row + i];
			break;

		case COL:
			if(C >= col) return;
			for(int i = 0; i < row; i++)
				c[i*col + C] = a[i*col + C] + b[i*col + C];
			break;

		case ELE:
			if(R >= row || C >= col) return;
			c[R*col + C] = a[R*col + C] + b[R*col + C];
			break;

		default:
			break;
	}
}

int main()
{
	int a[3][3] = {{1,2,3},{4,5,6},{7,8,9}};
	int b[3][3] = {{1,1,1},{1,1,1},{1,1,1}};
	int c[3][3];

	int *da, *db, *dc;
	int size = 3*3*sizeof(int);

	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size);

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, size, hipMemcpyHostToDevice);

	dim3 grid(1,1,1);
	dim3 block(5,5,1);

	MatrixAdd<<<grid, block>>>(da, db, dc, 3, 3, ELE);

	hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < 3; i++)
	{
		for(int j = 0; j < 3; j++)
			printf("%d  ", c[i][j]);
		printf("\n");
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}